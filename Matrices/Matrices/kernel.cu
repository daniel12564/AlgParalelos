
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <stdlib.h>

#define N 4096
#define T 1024

using namespace std;

__global__
void suma_vectores(int* a, int* b, int* c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
		c[i] = a[i] + b[i];
}

void llenar_vector(int* a)
{
	int i;
	for(i = 0; i < N; i++)
		a[i] = rand()%T;
}

void print_vector(int* a)
{
	int i;
	for(i = 0; i < N; i++)
		cout<<a[i]<<" ";
	cout<<endl;
}

int main()
{
	int a[N], b[N], c[N];
	llenar_vector(a);
	llenar_vector(b);
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);
	int tmp = ceil(N/T);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	suma_vectores<<<tmp, T>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size,hipMemcpyDeviceToHost);

	print_vector(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

    return 0;
}